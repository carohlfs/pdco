#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

// put the bandwidths in constant memory.
// note that max number cannot exceed 8*1024,
// given the 64kb limit for constant memory
__constant__ float d_bw[1000];

// GPU-specific implementation of quick-sort
// to be implemented by a single thread.
// Note that both A and B are sorted in
// increasing order of A.
// CUDA had problems with my recursive quick-sort,
// so I took an interative version from the
// following website.

// iterative version of quicksort taken
// from http://alienryderflex.com/quicksort/
// and then adapted to sort two arrays
// and to malloc the beg and end arrays.
__device__ void d_quicksort(float *A, float *B, int N) {

  #define  MAX_DEPTH  1000

  float pivot_A, pivot_B;
  int beg[MAX_DEPTH], end[MAX_DEPTH], i=0, L, R, swap;

  beg[0]=0; end[0]=N;
  while (i>=0) {
    L=beg[i]; R=end[i]-1;

    if (L<R) {
      pivot_A=A[L];
      pivot_B=B[L];
      
      while (L<R) {
        while (A[R]>=pivot_A && L<R) R--;
        if (L<R) {
          A[L]=A[R];
          B[L]=B[R];
          L++;
        }
        while (A[L]<=pivot_A && L<R) L++;
        if (L<R) {
          A[R]=A[L];
          B[R]=B[L];
          R--;
        }
      }
      
      A[L]=pivot_A;
      B[L]=pivot_B;

      beg[i+1]=L+1;
      end[i+1]=end[i];
      end[i++]=L;
      
      if (end[i]-beg[i]>end[i-1]-beg[i-1]) {
        swap=beg[i];
        beg[i]=beg[i-1];
        beg[i-1]=swap;
        swap=end[i];
        end[i]=end[i-1];
        end[i-1]=swap;
      }
    } else {
      i--;
    }
  }
}

// Here, we compute the Epanechnikov sum for a single
// xj value for an array of different bandwidths. The
// operations are performed simultaneously for multiple
// bandwidths to minimize the extent to which operations
// are duplicated.
__global__ void epanXYSum(float *yVec, float *xVec, float *xjxVec, float *yCopy, float *sumx, float *sumy, float *crossV, int N, int B){

	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if(j<N){

		int i,b;
		for(i=0;i<N;i++){
			xjxVec[N*j + i] = fabsf(xVec[j]-xVec[i]);
			yCopy[N*j + i] = yVec[i];
		}

		// sort the data so that the closest to xj
		// appear first. Then, if an observation i
		// is within some bandwidth b, then so is
		// every observation less than i. This
		// function also ensures that yCopy is sorted
		// in order of xjxVec.
		d_quicksort(&xjxVec[N*j], &yCopy[N*j], N);

		i=0;
		sumx[B*j] = 0.0;
		sumy[B*j] = 0.0;
		while(xjxVec[N*j+i]<=d_bw[0] && i<N){
			sumx[B*j]-=xjxVec[N*j+i]*xjxVec[N*j+i];
			sumy[B*j]-=yCopy[N*j+i]*xjxVec[N*j+i]*xjxVec[N*j+i];		
			i++;
		}

		for(b=1;b<B;b++){
			sumx[B*j+b]=sumx[B*j+b-1];
			sumy[B*j+b]=sumy[B*j+b-1];
			while(xjxVec[N*j+i]<=d_bw[b] && i<N){
				sumx[B*j+b]-=xjxVec[N*j+i]*xjxVec[N*j+i];
				sumy[B*j+b]-=yCopy[N*j+i]*xjxVec[N*j+i]*xjxVec[N*j+i];
				i++;
			}
		}

		for(b=0;b<B;b++){
			sumx[B*j+b]/=(d_bw[b]*d_bw[b]);
			sumx[B*j+b]=-0.75*sumx[B*j+b]; // note that the 0.75*1 is excluded due to leave-one-out.

			sumy[B*j+b]/=(d_bw[b]*d_bw[b]);
			sumy[B*j+b]=-0.75*sumy[B*j+b]; // similarly, we do not add 0.75*yVec[i] here.
		
			// switch the ordering of indices to facilitate the reduction.
			if(sumx[B*j+b]!=0.0){
				crossV[N*b+j]=(yVec[j] -sumy[B*j+b]/sumx[B*j+b])*(yVec[j] -sumy[B*j+b]/sumx[B*j+b])/N;
			} else {
				crossV[N*b+j]=FLT_MAX;
			}
		}
	}
}

// This function is taken from:
// http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
template <unsigned int blockSize>
__device__ void warpReduce_sum(volatile float *sdata, unsigned int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

// This function is a modified version of the reduce6 from:
// http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
template <unsigned int blockSize>
__global__ void reduce_sum(float *g_idata, float *g_odata, int n) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	sdata[tid] = 0.0;

	while (i < n) { sdata[tid] += g_idata[i]; i += blockSize; }
	__syncthreads();
	if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) warpReduce_sum<blockSize>(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// This function is modified from:
// http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
template <unsigned int blockSize>
__device__ void warpReduce_min(volatile float *sdata, unsigned int tid) {
	if (blockSize >= 64) {
		if(sdata[tid+32]<sdata[tid]){
			sdata[tid]=sdata[tid+32];
			sdata[tid+blockSize]=sdata[tid+32+blockSize];
		}
	};
	if (blockSize >= 32) {
		if(sdata[tid+16]<sdata[tid]){
			sdata[tid]=sdata[tid+16];
			sdata[tid+blockSize]=sdata[tid+16+blockSize];
		}
	}
	if (blockSize >= 16) {
		if(sdata[tid+8]<sdata[tid]){
			sdata[tid]=sdata[tid+8];
			sdata[tid+blockSize]=sdata[tid+8+blockSize];			
		}
	}
	if (blockSize >= 8) {
		if(sdata[tid+4]<sdata[tid]){
			sdata[tid]=sdata[tid+4];
			sdata[tid+blockSize]=sdata[tid+4+blockSize];			
		}	
	}
	if (blockSize >= 4) {
		if(sdata[tid+2]<sdata[tid]){
			sdata[tid]=sdata[tid+2];
			sdata[tid+blockSize]=sdata[tid+2+blockSize];			
		}
	}
	if (blockSize >= 2){
		if(sdata[tid+1]<sdata[tid]){
			sdata[tid]=sdata[tid+1];
			sdata[tid+blockSize]=sdata[tid+1+blockSize];			
		}
	}
}

// This function is a modified version of the reduce6 from:
// http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
template <unsigned int blockSize>
__global__ void reduce_min(float *g_idata, float *g_odata, int n) {
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	sdata[tid] = FLT_MAX;

	while (i < n) {
		if(g_idata[i]<sdata[tid]){
			sdata[tid] = g_idata[i];
			sdata[tid+blockSize]=d_bw[i];
		}
		i+=blockSize;
		__syncthreads();
	}

	if (blockSize >= 1024) {
		if (tid < 512) {
			if(sdata[tid+512]<sdata[tid]){
				sdata[tid]=sdata[tid+512];
				sdata[tid+blockSize]=sdata[tid+512+blockSize];
			}
		}
		__syncthreads();
	} 
	if (blockSize >= 512) {
		if (tid < 256) {
			if(sdata[tid+256]<sdata[tid]){
				sdata[tid]=sdata[tid+256];
				sdata[tid+blockSize]=sdata[tid+256+blockSize];
			}
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			if(sdata[tid+128]<sdata[tid]){
				sdata[tid]=sdata[tid+128];
				sdata[tid+blockSize]=sdata[tid+128+blockSize];
			}
			
		}
		__syncthreads();
	} 
	if (blockSize >= 128) {
		if (tid < 64) {
			if(sdata[tid+64]<sdata[tid]){
				sdata[tid]=sdata[tid+64];
				sdata[tid+blockSize]=sdata[tid+64+blockSize];
			}
		}
		__syncthreads();
	}
	if (tid < 32) warpReduce_min<blockSize>(sdata, tid);

	// only return the optimal bandwidth
	if (tid == 0) *g_odata = sdata[blockSize];
}

int main(int argc, char *argv[]) {

	srand(4251978);

	// constants used in calculations.
    // we'll allow user inputs but
    // have default values.
	int N=1024; // number of observations in vector
	int B = 50; // number of bandwidths in grid.
	const int T=512;

	if(argc>1){
		N = atoi(argv[1]);
	}

	int i;
	float *X = (float*)malloc(N*sizeof(float));
	float *Y = (float*)malloc(N*sizeof(float));

	for(i=0;i<N;i++){
		X[i] = (float)rand()/(float)RAND_MAX;
		Y[i] = 0.5*X[i] + 10*X[i]*X[i] + 0.5*(float)rand()/(float)RAND_MAX;
	}

	float minx=X[0], maxx=X[0];
	for(i=1;i<N;i++){
		if(X[i]<minx){
			minx = X[i];
		} else if(X[i]>maxx){
			maxx = X[i];
		}
	}

	float range = maxx -minx;

	if(argc>2){
		B = atoi(argv[2]);
	}
	if(argc>3){
		range = atof(argv[3]);
	}

	float increment = range/B;
	float start = increment;

	if(argc>4){
		start = atof(argv[4]);
		increment = range/(B-1);
	}

	int b;
	float *bw = (float*)malloc(B*sizeof(float));
	
	bw[0] = start;
	for(b=1;b<B;b++){
		bw[b] = bw[b-1] + increment;
	}

	float *d_X, *d_Y, *d_xjxVec, *d_yCopy, *d_sumx, *d_sumy, *d_crossV_elem;
	hipMalloc((void **)&d_X, N*sizeof(float));
	hipMalloc((void **)&d_Y, N*sizeof(float));
	hipMalloc((void **)&d_xjxVec, N*N*sizeof(float));
	hipMalloc((void **)&d_yCopy, N*N*sizeof(float));
	hipMalloc((void **)&d_sumx, B*N*sizeof(float));
	hipMalloc((void **)&d_sumy, B*N*sizeof(float));
	hipMalloc((void **)&d_crossV_elem, B*N*sizeof(float));

	hipMemcpy(d_X, X, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_bw), bw, B*sizeof(float));
	free(X);
	free(Y);

	hipError_t code = hipGetLastError();
	if(code!=hipSuccess){
        printf("Cuda error -- %s\n", hipGetErrorString(code));
    } 

	int numBlocks = N/T +1;
	dim3 threadsPerBlock(T);

	epanXYSum<<<numBlocks,threadsPerBlock>>>(d_Y,d_X,d_xjxVec,d_yCopy,d_sumx,d_sumy,d_crossV_elem,N,B);

	hipFree(d_sumx);
	hipFree(d_sumy);
	hipFree(d_Y);
	hipFree(d_X);

	// perform B separate reduction sums to obtain the
	// cross-validation statistic for each bandwidth.
	float *d_crossV_sum;
	hipMalloc((void **)&d_crossV_sum, B*sizeof(float));
	for(b=0;b<B;b++){
		reduce_sum<T><<< 1,threadsPerBlock,T*sizeof(float) >>>(&d_crossV_elem[N*b], &d_crossV_sum[b], N);
	}
	hipFree(d_crossV_elem);

	float *d_bw_optimal;
	hipMalloc((void **)&d_bw_optimal, sizeof(float));
	reduce_min<T><<< 1,threadsPerBlock,2*T*sizeof(float) >>>(d_crossV_sum, d_bw_optimal, B);
	hipFree(d_crossV_sum);

	float *bw_optimal = (float*)malloc(sizeof(float));
	hipMemcpy(bw_optimal, d_bw_optimal, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_bw_optimal);

	printf("Optimal bandwidth is: %f\n",*bw_optimal);
	printf("Min bandwidth is: %f\n",bw[0]);
	printf("Max bandwidth is: %f\n",bw[B-1]);
	free(bw);
	free(bw_optimal);

    return(0);
}